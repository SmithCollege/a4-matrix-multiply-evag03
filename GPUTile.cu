
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define SIZE 128
#define tileWi 2

 __global__ void MatrixMulOnDevice(float* A, float* B, float* C, int Width) {
  for (int i = 0; i < Width; ++i) {
    for (int j = 0; j < Width; ++j) {
      float sum = 0;
      for (int k = 0; k < Width; ++k) {
	float a = A[i * Width + k];
	float b = B[k * Width + j];
	sum += a * b;
      }
      C[i * Width + j] = sum;
    }
  }
}

int main() {
  int size = 100;

  float *x, *y, *z;
  hipMallocManaged(&x, SIZE*sizeof(float) * size * size);
  hipMallocManaged(&y, SIZE*sizeof(float) * size * size);
  hipMallocManaged(&z, SIZE*sizeof(float) * size * size);

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      x[i * size + j] = 1; // x[i][j]
      y[i * size + j] = 1; 
    }
    printf("\n");
  }

  MatrixMulOnDevice<<<pow((size/tileWi), 2.0), pow(tileWi, 2.0)>>>(x, y, z, size);

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipDeviceSynchronize();

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      printf("%f ", z[i * size + j]);
      if (z[i * size + j] != size) {
	printf("Error at z[%d][%d]: %f\n", i, j, z[i * size + j]);
      }
    }
    printf("\n");
  }
  
  hipFree(x);
  hipFree(y);
  hipFree(z);

  return 0;
}
