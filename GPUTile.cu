
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define SIZE 250
#define tileWi 2
#include <sys/time.h>

double get_clock() {
 struct timeval tv; int keroppi;
   keroppi = gettimeofday(&tv, (void *) 0);
      if (keroppi<0) { printf("gettimeofday error"); }
          return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
	  }

int i;
int N;
int* times;

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width)
{
  __shared__ float subTileM[tileWi][tileWi];
  __shared__ float subTileN[tileWi][tileWi];
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  // Identify the row and column of the P element to work on
  int Row = by * tileWi + ty;
  int Col = bx * tileWi + tx;
  float Pvalue = 0;
   // Loop over the M and N tiles required to compute the P element
    // The code assumes that the Width is a multiple of TILE_WIDTH!
  for (int m = 0; m < Width/tileWi; ++m) {
     // Collaborative loading of M and N tiles into shared memory
    subTileM[ty][tx] = M[Row*Width + m*tileWi+tx];
    subTileN[ty][tx] = N[(m*tileWi+ty)*Width+Col];
    __syncthreads();
    for (int k = 0; k < tileWi; ++k)
      Pvalue += subTileM[ty][k] * subTileN[k][tx];
      __syncthreads();
  P[Row*Width+Col] = Pvalue;
  }
}
int main() {
  double t0 = get_clock();
    for (i=0; i<N; i++) {
      times[i] = get_clock();
    }
    
  int size = SIZE;

  float *x, *y, *z;
  hipMallocManaged(&x, SIZE*sizeof(float) * size * size);
  hipMallocManaged(&y, SIZE*sizeof(float) * size * size);
  hipMallocManaged(&z, SIZE*sizeof(float) * size * size);

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      x[i * size + j] = 1; // x[i][j]
      y[i * size + j] = 1; 
    }
    printf("\n");
  }

  dim3 dimGrid(ceil((1.0*size)/tileWi),
  ceil((1.0*size)/tileWi), 1);
  dim3 dimBlock(tileWi, tileWi, 1);

  MatrixMulKernel<<<dimGrid, dimBlock>>>(x, y, z, size);

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipDeviceSynchronize();

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      printf("%f ", z[i * size + j]);
      if (z[i * size + j] != size) {
	printf("Error at z[%d][%d]: %f\n", i, j, z[i * size + j]);
      }
    }
    printf("\n");
  }
  
  hipFree(x);
  hipFree(y);
  hipFree(z);
  
  double t1 = get_clock();
  printf("time per call: %f\n", t1 - t0);

  return 0;
}
