
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define SIZE 250
#include <sys/time.h>

double get_clock() {
 struct timeval tv; int keroppi;
   keroppi = gettimeofday(&tv, (void *) 0);
      if (keroppi<0) { printf("gettimeofday error"); }
          return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
	  }

int i;
int N;
int* times;

 __global__ void MatrixMulOnDevice(float* A, float* B, float* C, int Width) {
  for (int i = 0; i < Width; ++i) {
    for (int j = 0; j < Width; ++j) {
      float sum = 0;
      for (int k = 0; k < Width; ++k) {
	float a = A[i * Width + k];
	float b = B[k * Width + j];
	sum += a * b;
      }
      C[i * Width + j] = sum;
    }
  }
}

int main() {

  double t0 = get_clock();
    for (i=0; i<N; i++) {
      times[i] = get_clock();
    }

  int size = SIZE;

  float *x, *y, *z;
  hipMallocManaged(&x, SIZE*sizeof(float) * size * size);
  hipMallocManaged(&y, SIZE*sizeof(float) * size * size);
  hipMallocManaged(&z, SIZE*sizeof(float) * size * size);

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      x[i * size + j] = 1; // x[i][j]
      y[i * size + j] = 1; 
    }
    printf("\n");
  }

  MatrixMulOnDevice<<<1, 128>>>(x, y, z, size);

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipDeviceSynchronize();

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      printf("%f ", z[i * size + j]);
      if (z[i * size + j] != size) {
	printf("Error at z[%d][%d]: %f\n", i, j, z[i * size + j]);
      }
    }
    printf("\n");
  }
  
  hipFree(x);
  hipFree(y);
  hipFree(z);

  double t1 = get_clock();
  printf("time per call: %f\n", t1 - t0);
   
  return 0;
}
